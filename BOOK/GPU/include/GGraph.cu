#include "hip/hip_runtime.h"
#ifndef GGRAPH_H
#define GGRAPH_H
#include "Graph.h"
#include <stdio.h>
#include </usr/local/cuda/include/hip/hip_runtime.h>
    #include </usr/local/cuda/include/hip/hip_runtime_api.h>
class HGraph;
class GGraph: public Graph {
public:
//int *maxnpoints,*pnpoints,*maxnedges,*pnedges;
int maxnpoints,*pnpoints,maxnedges,*pnedges;//made this as only pnpoints and pnedges are used in atomic.
int allocate();
 int copytoCPU(HGraph &hgraph);
int  copyfromCPU(HGraph &hgraph);
 int cloneCPU(HGraph &hgraph);
 int deallocate();
//__device__  int addPoint(int x,int y);
//__constant__ union float_int   *edges;
__device__ int getEdgeWeight(int t1,int t2);
__device__ int addPoint(float x,float y);
__device__ int addPoint(int x,int y);
__device__ int addEdge(int p1,int p2);
__device__ int addEdge(int p1,int p2,int weight);
 void (*extra_alloc)(GGraph &);
};
__device__ int GGraph::addPoint(float x,float y){
int x1=1;
int t1;
t1=atomicAdd(pnpoints,x1);
if(t1<(maxnpoints)){
points[2*t1].fpe=x;
points[2*t1+1].fpe=y;
}
return t1;
}
__device__ int GGraph::addPoint(int x,int y){
int x1=1;
int t1;
t1=atomicAdd(pnpoints,x1);
if(t1<(maxnpoints)){
points[2*t1].ipe=x;
points[2*t1+1].ipe=y;
}
return t1;
}
__device__ int GGraph::addEdge(int p1,int p2){
//int x=1;
int t1;
t1=atomicAdd(&total[p1],1);
edges[2*(index[p1]+t1)].ipe=p2;
edges[2*(index[p1]+t1)+1].ipe=0;
return 1;
}
__device__ int GGraph::addEdge(int p1,int p2,int weight){
//int x=1;
int t1;
t1=atomicAdd(&total[p1],1);
edges[2*(index[p1]+t1)].ipe=p2;
edges[2*(index[p1]+t1)+1].ipe=weight;
return 1;
}
//__device__
 volatile unsigned int *arrayin,*arrayout;
__device__ volatile unsigned int ARRU[1024],ARRO[1024];
unsigned int goal;
void alloc_sync_array(){
	if (hipMalloc((void **)&arrayin, 1024*sizeof(volatile unsigned int )) != hipSuccess) printf("allocating arrayin failed");
	if (hipMalloc((void **)&arrayout, 1024*sizeof(volatile unsigned int )) != hipSuccess) printf("allocating arrayout failed");
}
__device__ 
void __gpu_sync(unsigned goalVal, volatile unsigned *Arrayin, volatile unsigned *Arrayout) {
	// thread ID in a block
	unsigned tid_in_blk = threadIdx.x * blockDim.y + threadIdx.y;
	unsigned nBlockNum = gridDim.x * gridDim.y;
	unsigned bid = blockIdx.x * gridDim.y + blockIdx.y;
	// only thread 0 is used for synchronization
	if (tid_in_blk == 0) {
		Arrayin[bid] = goalVal;
		__threadfence();
	}
	if (bid == 0) {
		if (tid_in_blk < nBlockNum) {
			while (Arrayin[tid_in_blk] != goalVal){
				//Do nothing here
			}
		}
		__syncthreads();
		if (tid_in_blk < nBlockNum) {
			Arrayout[tid_in_blk] = goalVal;
			__threadfence();
		}
	}
	if (tid_in_blk == 0) {
		while (Arrayout[bid] != goalVal) {
			//Do nothing here
		}
	}
	__syncthreads();
}
#endif
