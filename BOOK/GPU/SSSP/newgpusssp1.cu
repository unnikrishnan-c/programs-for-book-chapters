#include "hip/hip_runtime.h"
#include "newgpusssp1.h"

hipDeviceProp_t prop0;
__device__ int changed = 0;;
int hchanged = 0;

__global__ void relaxgraph(GGraph graph, int FALCX)
{
	int id = blockIdx.x *blockDim.x + threadIdx.x + FALCX;

	int p = id;
	if (id < graph.npoints && ((struct struct_hgraph *)(graph.extra))->/*4*/ updated[id] == true)
	{
		((struct struct_hgraph *)(graph.extra))->updated[id] = false;

		int falcft0 = graph.index[id + 1] - graph.index[id];
		int falcft1 = graph.index[id];
		for (int falcft2 = 0; falcft2 < falcft0; falcft2++)
		{
			int ut0 = 2 *(falcft1 + falcft2);
			int ut1 = graph.edges[ut0].ipe;
			int ut2 = graph.edges[ut0 + 1].ipe;
			int t = ut1;
			GMIN(&(((struct struct_hgraph *)(graph.extra))->dist[ut1]), ((struct struct_hgraph *)(graph.extra))->dist[id] + ut2, changed);
		}
	}
}

__global__ void reset(GGraph graph, int FALCX)
{
	int id = blockIdx.x *blockDim.x + threadIdx.x + FALCX;

	int t = id;
	if (id < graph.npoints)
	{
		((struct struct_hgraph *)(graph.extra))->dist[id] = 1234567890;

		((struct struct_hgraph *)(graph.extra))->olddist[id] = 1234567890;

		((struct struct_hgraph *)(graph.extra))->updated[id] = false;
	}
}

__global__ void reset1(GGraph graph, int FALCX)
{
	int id = blockIdx.x *blockDim.x + threadIdx.x + FALCX;

	int t = id;
	if (id < graph.npoints)
	{
		if (((struct struct_hgraph *)(graph.extra))->dist[id] < ((struct struct_hgraph *)(graph.extra))->olddist[id])
			((struct struct_hgraph *)(graph.extra))->updated[id] = true;

		((struct struct_hgraph *)(graph.extra))->olddist[id] = ((struct struct_hgraph *)(graph.extra))->dist[id];
	}
}

void SSSP(char *name)
{
	HGraph hgraph;

	GGraph graph;

	hgraph.read2(name);

	int hosthgraph = 0;
	alloc_extra_hgraph(hgraph, hosthgraph, hgraph.npoints);;

	hgraph.cloneGPU(graph, 0);
	int graphflag = 0;

	alloc_extra_graph(graph, graphflag, graph.npoints);

	int TPB0 = 1024;

	//int graphpointkernelblocks=findblocksize(graph,graph.npoints,TPB0);

	//int graphedgekernelblocks=findblocksize(graph,graph.nedges,TPB0);
	int graphpointkernelblocks = graph.npoints / 1024 + 1;
	copygraphcurrentsize(graph);
	hipSetDevice(0);

	double t1, t2;

	t1 = rtclock();

	hipSetDevice(0);
	for (int kk = 0; kk < graph.npoints; kk += graphpointkernelblocks *TPB0)
	{
		reset<<<graphpointkernelblocks, TPB0>>> (graph, kk);
	}

	hipDeviceSynchronize();
	hipSetDevice(0);

	bool falcvt1;
	falcvt1 = true;
	struct struct_hgraph temp1;
	hipMemcpy(&temp1, ((struct struct_hgraph *)(graph.extra)), sizeof(struct struct_hgraph), hipMemcpyDeviceToHost);
	if (hipMemcpy(&(temp1.updated[0]), &(falcvt1), sizeof(bool), hipMemcpyHostToDevice) != hipSuccess) printf("memcpyerror 1");

	int falcvt2;
	falcvt2 = 0;
	struct struct_hgraph temp2;
	hipMemcpy(&temp2, ((struct struct_hgraph *)(graph.extra)), sizeof(struct struct_hgraph), hipMemcpyDeviceToHost);
	if (hipMemcpy(&(temp2.dist[0]), &(falcvt2), sizeof(int), hipMemcpyHostToDevice) != hipSuccess) printf("memcpyerror 2");

	while (1)
	{
		int falcvt3;
		falcvt3 = 0;
		if (hipMemcpyToSymbol(HIP_SYMBOL(changed), &(falcvt3), sizeof(int), 0, hipMemcpyHostToDevice) != hipSuccess) printf("memcpyerror 3");

		hipSetDevice(0);
		for (int kk = 0; kk < graph.npoints; kk += graphpointkernelblocks *TPB0)
		{
			relaxgraph <<<graphpointkernelblocks, TPB0>>> (graph, kk);
		}

		hipDeviceSynchronize();
		hipSetDevice(0);

		int falcvt4;
		if (hipMemcpyFromSymbol(&(falcvt4), changed, sizeof(int), 0, hipMemcpyDeviceToHost) != hipSuccess) printf("memcpyerror 4");
		if (falcvt4 == 0) break;

		hipSetDevice(0);
		for (int kk = 0; kk < graph.npoints; kk += graphpointkernelblocks *TPB0)
		{
			reset1 <<<graphpointkernelblocks, TPB0>>> (graph, kk);
		}

		hipDeviceSynchronize();
		hipSetDevice(0);
	}

	t2 = rtclock();

	sleep(5);
	printf("\nTIME =%f", (t2 - t1) *1000);

	struct struct_hgraph temp3; /*err*/
	hipMemcpy(&temp3, ((struct struct_hgraph *)(graph.extra)), sizeof(struct struct_hgraph), hipMemcpyDeviceToHost);
	if (hipMemcpy((((struct struct_hgraph *)(hgraph.extra))->dist), (temp3.dist), sizeof(int) *hgraph.npoints, hipMemcpyDeviceToHost) != hipSuccess) printf("memcpyerror 5");

	int maxdist = 0;

	for (int i = 0; i < hgraph.npoints; i++)
	{
		if (maxdist < ((struct struct_hgraph *)(hgraph.extra))->dist[i])
			maxdist = ((struct struct_hgraph *)(hgraph.extra))->dist[i];
	}

	printf("\nMAXDIST=%d\n", maxdist);

	return;

}

int main(int argc, char *argv[])
{
	hipGetDeviceProperties(&prop0, 0);
	//alloc_sync_array();

	if (argc != 2)
	{
		printf("error: exec filename");

		return 1;
	}

	SSSP(argv[1]);

}
