#include "hip/hip_runtime.h"

 #include "undirected_triangle_counting.h"
hipDeviceProp_t prop0;
__device__ int   changed =0;
;
 int   hchanged ;


 __global__ void   TC ( GGraph  graph ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int p=id;
if( id < graph.npoints){

 struct dev_vec coll;
coll= create_coll_device(100);


  int   ch ;


 int falcft0=graph.index[id+1]-graph.index[id];
int falcft1=graph.index[id];
for(int falcft2=0;falcft2<falcft0;falcft2++){
int ut0=2*(falcft1+falcft2);
 int ut1=graph.edges[ut0].ipe;
int ut2=graph.edges[ut0+1].ipe;
int t=ut1;
int t1;

 t1=t; 

 if( graph.points[t1].ipe<=graph.points[id].ipe )
continue;

 add_coll_device(coll,t1); 

 }

 for (int   i =0;i<coll.size;i++) {

 int p2;

 p2=coll.D_Vec[i]; 

 int falcft3=graph.index[p2+1]-graph.index[p2];
int falcft4=graph.index[p2];
for(int falcft5=0;falcft5<falcft3;falcft5++){
int ut3=2*(falcft4+falcft5);
 int ut4=graph.edges[ut3].ipe;
int ut5=graph.edges[ut3+1].ipe;
int t2=ut4;
int t1;

 t1=t2; 

 if( graph.points[t1].ipe<=graph.points[p2].ipe )
continue;

 for (int   i =0;i<coll.size;i++) {

 int p3;

 p3=coll.D_Vec[i]; 

 if( graph.points[p3].ipe==graph.points[t1].ipe )
GADD(&(changed),1);


 }

 }

 }

 }

}
int   main ( int   argc ,char    *  name [ ] ) 
 {
hipGetDeviceProperties(&prop0,0); 
alloc_sync_array();


 if( argc!=2  )
{

 printf("error: exec filename");


 return 1;

 }

 HGraph  hgraph ;

 

 hgraph.read2(name[1]);

int hosthgraph=0;
alloc_extra_hgraph(hgraph,hosthgraph,hgraph.npoints);
; 

 GGraph graph;


 hgraph.cloneGPU(graph,0 );
int graphflag=0;

alloc_extra_graph(graph,graphflag,graph.npoints);

int TPB0=findthreadsperblock(prop0);

int graphpointkernelblocks=findblocksize(graph,graph.npoints,TPB0);

int graphedgekernelblocks=findblocksize(graph,graph.nedges,TPB0);
copygraphcurrentsize(graph);
hipSetDevice(0);


  double   t1 , t2 ;


 t1=rtclock();


 int   falcvt1;
falcvt1=0;
struct struct_hgraph  temp1;
 hipMemcpy(&temp1,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
if(hipMemcpy(&(temp1.dist[0]),&(falcvt1),sizeof(int ),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 1");

 int   falcvt2;
falcvt2=0;
if(hipMemcpyToSymbol(HIP_SYMBOL(changed),&(falcvt2),sizeof(int ),0,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 2");

 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
TC<<<graphpointkernelblocks,TPB0>>>(graph,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);


 t2=rtclock();


 printf("\nTIME =%f\n",(t2-t1)*1000);


  int   count ;


 if(hipMemcpyFromSymbol(&(count),changed,sizeof(int ),0,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 3");

 printf("\ntriangles=%d \n",count);


 return ;

 }
