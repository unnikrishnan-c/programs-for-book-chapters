#include "hip/hip_runtime.h"

 #include "levelbfs.h"
hipDeviceProp_t prop0;
__device__ int   changed =0;
;
 int   hchanged ;


  int   lev =0;


 __global__ void   reset ( GGraph  graph ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int t=id;
if( id < graph.npoints){

 ((struct struct_hgraph  *)(graph.extra))->dist[id]=1234567890; 

 }

}
__global__ void   BFS ( GGraph  graph ,int   lev ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int p=id;
if( id < graph.npoints&& ((struct struct_hgraph  *)(graph.extra))->/*4*/dist[id]==lev ){

  int   ch ;


 int falcft0=graph.index[id+1]-graph.index[id];
int falcft1=graph.index[id];
for(int falcft2=0;falcft2<falcft0;falcft2++){
int ut0=2*(falcft1+falcft2);
 int ut1=graph.edges[ut0].ipe;
int ut2=graph.edges[ut0+1].ipe;
int t=ut1;
if( ((struct struct_hgraph  *)(graph.extra))->dist[ut1]>(((struct struct_hgraph  *)(graph.extra))->dist[id]+1) )
{

 ((struct struct_hgraph  *)(graph.extra))->dist[ut1]=((struct struct_hgraph  *)(graph.extra))->dist[id]+1; 

 changed=1; 

 }

 }

 }

}
int   main ( int   argc ,char    *  name [ ] ) 
 {
hipGetDeviceProperties(&prop0,0); 
//alloc_sync_array();


 if( argc!=2  )
{

 printf("error: exec filename");


 return 1;

 }

 HGraph  hgraph ;

 

 hgraph.read2(name[1]);

int hosthgraph=0;
alloc_extra_hgraph(hgraph,hosthgraph,hgraph.npoints);
; 

 GGraph graph;


 hgraph.cloneGPU(graph,0 );
int graphflag=0;

alloc_extra_graph(graph,graphflag,graph.npoints);

int TPB0=1024;

int graphpointkernelblocks=graph.npoints/1024+1;

//int graphedgekernelblocks=findblocksize(graph,graph.nedges,TPB0);
copygraphcurrentsize(graph);
hipSetDevice(0);


  double   t1 , t2 ;


 t1=rtclock();


 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
reset<<<graphpointkernelblocks,TPB0>>>(graph,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);


 int   falcvt1;
falcvt1=0;
struct struct_hgraph  temp1;
 hipMemcpy(&temp1,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
if(hipMemcpy(&(temp1.dist[2]),&(falcvt1),sizeof(int ),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 1");

 
 while(1)  { 

 int   falcvt2;
falcvt2=0;
if(hipMemcpyToSymbol(HIP_SYMBOL(changed),&(falcvt2),sizeof(int ),0,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 2");

 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
BFS<<<graphpointkernelblocks,TPB0>>>(graph,lev,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);
sleep(1);

 int   falcvt3;
if(hipMemcpyFromSymbol(&(falcvt3),changed,sizeof(int ),0,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 3");
if(falcvt3==0)break;

 lev++; 

 }

 t2=rtclock();


 printf("\nTIME =%f\n",(t2-t1)*1000);


 struct struct_hgraph  temp2;/*err*/
hipMemcpy(&temp2,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
if(hipMemcpy((((struct struct_hgraph  *)(hgraph.extra))->dist),(temp2.dist),sizeof(int)*hgraph.npoints,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 4");

  int   maxdist =0;


 for (int   i =0;i<hgraph.npoints;i++) {

 if( (maxdist<((struct struct_hgraph  *)(hgraph.extra))->dist[i])&&( ((struct struct_hgraph  *)(hgraph.extra))->dist[i] < 10000)  )
maxdist=((struct struct_hgraph  *)(hgraph.extra))->dist[i]; 

 }

 printf("\nMAXDIST=%d \n",maxdist);


 return ;

 }
