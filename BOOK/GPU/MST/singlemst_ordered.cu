#include "hip/hip_runtime.h"

 #include "singlemst_ordered.h"
hipDeviceProp_t prop0;


 __device__ int   changed ;
;
 int   hchanged ;


 __global__ void   reset ( GGraph  graph ,/*SET*/GSet set ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int p=id;
if( id < graph.npoints){

 ((struct struct_hgraph  *)(graph.extra))->minppty[id].weight=99999999; 

 ((struct struct_hgraph  *)(graph.extra))->minedge[id]=99999999; 

 ((struct struct_hgraph  *)(graph.extra))->minppty[id].set=99999999; 

 ((struct struct_hgraph  *)(graph.extra))->minppty[id].lock=0; 

 }

}
__global__ void   minset ( GGraph  graph ,/*SET*/GSet set ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int p=id;
if( id < graph.npoints){

  int   ch ;


 int t1;

 int t2;

 int falcft0=graph.index[id+1]-graph.index[id];
int falcft1=graph.index[id];
for(int falcft2=0;falcft2<falcft0;falcft2++){
int ut0=2*(falcft1+falcft2);
 int ut1=graph.edges[ut0].ipe;
int ut2=graph.edges[ut0+1].ipe;
int t=ut1;
t1=set.parent[id]; 

 t2=set.parent[ut1]; 

 if( t1!=t2 )
{

 GMIN(&(((struct struct_hgraph  *)(graph.extra))->minppty[t1].weight),ut2,ch);


 GMIN(&(((struct struct_hgraph  *)(graph.extra))->minppty[t2].weight),ut2,ch);


 }

 }

 }

}
__global__ void   Minedge ( GGraph  graph ,/*SET*/GSet set ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int p=id;
if( id < graph.npoints){

 int t1;

 int t2;

 int tt2;

  int   t3 ;


 int e;

 int falcft3=graph.index[id+1]-graph.index[id];
int falcft4=graph.index[id];
for(int falcft5=0;falcft5<falcft3;falcft5++){
int ut3=2*(falcft4+falcft5);
 int ut4=graph.edges[ut3].ipe;
int ut5=graph.edges[ut3+1].ipe;
int t=ut4;
t1=set.parent[id]; 

 t2=set.parent[ut4]; 

 if( graph.points[t1].ipe>graph.points[t2].ipe )
{

 tt2=t2; 

 t2=t1; 

 t1=tt2; 

 }

 t3=ut5; 

 if( t1!=t2 )
{

 if( t3==((struct struct_hgraph  *)(graph.extra))->minppty[t1].weight )
{

 if(atomicCAS( &(((struct struct_hgraph  *)(graph.extra))->minppty[t1].lock),0,1)==0){

 e=ut3/2; 

 ((struct struct_hgraph  *)(graph.extra))->mark[e]=1; 

 ((struct struct_hgraph  *)(graph.extra))->minppty[t1].src=p; 

 ((struct struct_hgraph  *)(graph.extra))->minppty[t1].dst=t; 

 ((struct struct_hgraph  *)(graph.extra))->minppty[t1].weight=t3; 

 ((struct struct_hgraph  *)(graph.extra))->minppty[t1].set=t2; 

 }

 }

 if( t3==((struct struct_hgraph  *)(graph.extra))->minppty[t2].weight )
{

 if(atomicCAS( &(((struct struct_hgraph  *)(graph.extra))->minppty[t2].lock),0,1)==0){

 e=ut3/2; 

 ((struct struct_hgraph  *)(graph.extra))->mark[e]=1; 

 ((struct struct_hgraph  *)(graph.extra))->minppty[t2].src=p; 

 ((struct struct_hgraph  *)(graph.extra))->minppty[t2].dst=t; 

 ((struct struct_hgraph  *)(graph.extra))->minppty[t2].weight=t3; 

 ((struct struct_hgraph  *)(graph.extra))->minppty[t2].set=t1; 

 }

 }

 }

 }

 }

}
__global__ void   mstunion ( GGraph  graph ,/*SET*/GSet set ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int p=id;
if( id < graph.npoints){

 int t1;

 int t2;

  int   t3 , t4 ;


 t1=set.parent[id]; 

 t2=((struct struct_hgraph  *)(graph.extra))->minppty[t1].set; 

 t3=((struct struct_hgraph  *)(graph.extra))->minppty[t1].lock; 

 t4=((struct struct_hgraph  *)(graph.extra))->minppty[t2].lock; 

 if( t1!=t2&&t3==1 )
{

 //Currn_fun_name =mstunion
set.Union(t1,t2); 

 changed=1; 

 }

 }

}
__global__ void   initmark ( GGraph  graph ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int e=id;
if( id < graph.nedges){

 ((struct struct_hgraph  *)(graph.extra))->mark[id]=999999999; 

 }

}
int   main ( int   argc ,char    *  argv [ ] ) 
 {
hipGetDeviceProperties(&prop0,0); 
alloc_sync_array();


 if( argc!=2  )
{

 printf("error:- exec filename");


 return 1;

 }

 HGraph  hgraph ;

 

 

 

 GGraph graph;


 hgraph.read2(argv[1]);

int hosthgraph=0;
alloc_extra_hgraph(hgraph,hosthgraph,hgraph.npoints);
; 

 HSet hset;
hset.allocate(hgraph.npoints);

 GSet set;
set.allocate(graph.npoints);

 hgraph.cloneGPU(graph,0 );
int graphflag=0;

alloc_extra_graph(graph,graphflag,graph.npoints);

int TPB0=findthreadsperblock(prop0);

int graphpointkernelblocks=findblocksize(graph,graph.npoints,TPB0);

int graphedgekernelblocks=findblocksize(graph,graph.nedges,TPB0);
copygraphcurrentsize(graph);
hipSetDevice(0);


 hset.cloneGPU(set,0 );
int setflag=0;
hipSetDevice(0);


 hchanged=0; 

  double   rt1 =rtclock();


 hipSetDevice(0);
for(int kk=0;kk<graph.nedges;kk+=graphedgekernelblocks*TPB0){ 
initmark<<<graphedgekernelblocks,TPB0>>>(graph,kk);}
hipDeviceSynchronize();
hipSetDevice(0);


 
 while(1)  { 

 int   falcvt1;
falcvt1=0;
if(hipMemcpyToSymbol(HIP_SYMBOL(changed),&(falcvt1),sizeof(int ),0,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 1");

 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
reset<<<graphpointkernelblocks,TPB0>>>(graph,set,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);


 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
minset<<<graphpointkernelblocks,TPB0>>>(graph,set,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);


 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
Minedge<<<graphpointkernelblocks,TPB0>>>(graph,set,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);


 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
mstunion<<<graphpointkernelblocks,TPB0>>>(graph,set,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);
hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){
findset<<<graphpointkernelblocks,TPB0>>>( set,kk);
}
hipDeviceSynchronize();hipSetDevice(0);


 int   falcvt2;
if(hipMemcpyFromSymbol(&(falcvt2),changed,sizeof(int ),0,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 2");
if(falcvt2==0)break;

 }

  double   rt2 =rtclock();


 printf("TIME=%f \n",(rt2-rt1)*1000);


 struct struct_hgraph  temp2;/*err*/
hipMemcpy(&temp2,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
if(hipMemcpy((((struct struct_hgraph  *)(hgraph.extra))->mark),(temp2.mark),sizeof(int)*hgraph.nedges,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 3");

  unsigned  long  int   mst =0;


 unsigned int hreduxsum0=0;
hipMemcpyToSymbol(HIP_SYMBOL(reduxsum0),&hreduxsum0,sizeof(unsigned int ),0,hipMemcpyHostToDevice);
hipSetDevice(0);
if(graph.nedges/1024> (32*1024)){
for(int kk=0;kk<graph.nedges;kk+=FALCK32*1024)RSUM0<<<FALCK32,1024>>>(graph,kk);
hipDeviceSynchronize();
hipSetDevice(0);}

if(graph.nedges/1024< (32*1024)){
RSUM0<<<graph.nedges/1024+1,1024>>>(graph,0);
hipDeviceSynchronize();
hipSetDevice(0);}
hipMemcpyFromSymbol(&hreduxsum0,HIP_SYMBOL(reduxsum0),sizeof(unsigned int ),0,hipMemcpyDeviceToHost);
mst=hreduxsum0;
printf("\nmst cost=%lu\n",mst);


 }
