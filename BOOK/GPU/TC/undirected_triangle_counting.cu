#include "hip/hip_runtime.h"
#include "undirected_triangle_counting.h"

hipDeviceProp_t prop0;
__device__ int changed = 0;;
int hchanged;
__global__ void TC(GGraph graph, int FALCX)
{ //each traingle counted six times, naive algorithm
	int id = blockIdx.x *blockDim.x + threadIdx.x + FALCX;

	if (id < graph.npoints)
	{
		int p = id;//p
		int startsrc=graph.index[p];
		int endsrc=graph.index[p+1];
		for(int kk=startsrc;kk<endsrc;kk++){
			int t=graph.edges[2*kk].ipe;//t
			for(int ll=startsrc;ll<endsrc;ll++){//
				int r=graph.edges[2*ll].ipe;//r, edges p--t, p--r
				if(t==r)continue;//t==r go to next edge.
				int nbrs_r_start=graph.index[r];//check edge t--r exist, where t!=r
				int nbrs_r_end=graph.index[r+1];
				for(int i=nbrs_r_start;i<nbrs_r_end;i++)
				       	if (graph.edges[2*i].ipe==t)atomicAdd(&changed,1);//edge t--r exist, making triangle p-t,t--r, t--p
			}
		}
	}
}

int main(int argc, char *name[])
{
	hipGetDeviceProperties(&prop0, 0);
	alloc_sync_array();

	if (argc != 2)
	{
		printf("error: exec filename");

		return 1;
	}

	HGraph hgraph;

	hgraph.read2(name[1]);

	int hosthgraph = 0;
	alloc_extra_hgraph(hgraph, hosthgraph, hgraph.npoints);;

	GGraph graph;

	hgraph.cloneGPU(graph, 0);
	int graphflag = 0;

	alloc_extra_graph(graph, graphflag, graph.npoints);

	int TPB0 = 1024;

	int graphpointkernelblocks = graph.npoints/ TPB0+1;

	int graphedgekernelblocks = graph.nedges/ TPB0+1;
	copygraphcurrentsize(graph);
	hipSetDevice(0);

	double t1, t2;

	t1 = rtclock();

	int falcvt1;
	falcvt1 = 0;
	struct struct_hgraph temp1;
	hipMemcpy(&temp1, ((struct struct_hgraph *)(graph.extra)), sizeof(struct struct_hgraph), hipMemcpyDeviceToHost);
	if (hipMemcpy(&(temp1.dist[0]), &(falcvt1), sizeof(int), hipMemcpyHostToDevice) != hipSuccess) printf("memcpyerror 1");

	int falcvt2;
	falcvt2 = 0;
	if (hipMemcpyToSymbol(HIP_SYMBOL(changed), &(falcvt2), sizeof(int), 0, hipMemcpyHostToDevice) != hipSuccess) printf("memcpyerror 2");
printf("%d %d %d\n",  graph.npoints, graphpointkernelblocks,TPB0);
	hipSetDevice(0);
	for (int kk = 0; kk < graph.npoints; kk += graphpointkernelblocks *TPB0)
	{
		TC <<<graphpointkernelblocks, TPB0>>> (graph, kk);
	}

	hipDeviceSynchronize();
	hipSetDevice(0);

	t2 = rtclock();

	printf("\nTIME =%f\n", (t2 - t1) *1000);

	int count;

	if (hipMemcpyFromSymbol(&(count), changed, sizeof(int), 0, hipMemcpyDeviceToHost) != hipSuccess) printf("memcpyerror 3");
        int triangles=count/6;
	printf("\ntriangles=%d \n", triangles);

	return;

}
