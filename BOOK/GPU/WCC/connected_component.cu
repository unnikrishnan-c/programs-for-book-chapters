#include "hip/hip_runtime.h"

 #include "connected_component.h"
hipDeviceProp_t prop0;
__device__ int   changed =0;
;
 int   hchanged =0;


 __global__ void   relaxgraph ( GGraph  unnigraph ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int p=id;
if( id < unnigraph.npoints){

 int falcft0=unnigraph.index[id+1]-unnigraph.index[id];
int falcft1=unnigraph.index[id];
for(int falcft2=0;falcft2<falcft0;falcft2++){
int ut0=2*(falcft1+falcft2);
 int ut1=unnigraph.edges[ut0].ipe;
int ut2=unnigraph.edges[ut0+1].ipe;
int t=ut1;
GMIN(&(((struct struct_hgraph  *)(unnigraph.extra))->dist[ut1]),((struct struct_hgraph  *)(unnigraph.extra))->dist[id],changed);


 GMIN(&(((struct struct_hgraph  *)(unnigraph.extra))->dist[id]),((struct struct_hgraph  *)(unnigraph.extra))->dist[ut1],changed);


 }

 }

}
__global__ void   reset ( GGraph  graph ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int t=id;
if( id < graph.npoints){

 ((struct struct_hgraph  *)(graph.extra))->dist[id]=graph.points[id].ipe; 

 }

}
void   SSSP ( char    *  name ) 
 {

 HGraph  hgraph ;

 

 GGraph graph;


 hgraph.read2(name);

int hosthgraph=0;
alloc_extra_hgraph(hgraph,hosthgraph,hgraph.npoints);
; 

 hgraph.cloneGPU(graph,0 );
int graphflag=0;

alloc_extra_graph(graph,graphflag,graph.npoints);

int TPB0=1024;

int graphpointkernelblocks=graph.npoints/1024+1;

copygraphcurrentsize(graph);
hipSetDevice(0);


  double   t1 , t2 ;


 t1=rtclock();


 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
reset<<<graphpointkernelblocks,TPB0>>>(graph,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);


 int   falcvt1;
falcvt1=0;
struct struct_hgraph  temp1;
 hipMemcpy(&temp1,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
if(hipMemcpy(&(temp1.dist[0]),&(falcvt1),sizeof(int ),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 1");

 
 while(1)  { 

 int   falcvt2;
falcvt2=0;
if(hipMemcpyToSymbol(HIP_SYMBOL(changed),&(falcvt2),sizeof(int ),0,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 2");

 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
relaxgraph<<<graphpointkernelblocks,TPB0>>>(graph,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);


 int   falcvt3;
if(hipMemcpyFromSymbol(&(falcvt3),changed,sizeof(int ),0,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 3");
if(falcvt3==0)break;

 }

 t2=rtclock();


 printf("\nTIME =%f",(t2-t1)*1000);


 struct struct_hgraph  temp2;/*err*/
hipMemcpy(&temp2,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
if(hipMemcpy((((struct struct_hgraph  *)(hgraph.extra))->dist),(temp2.dist),sizeof(int)*hgraph.npoints,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 4");

  int   maxdist =0;


 for (int   i =0;i<hgraph.npoints;i++) {

 if( maxdist<((struct struct_hgraph  *)(hgraph.extra))->dist[i]  )
maxdist=((struct struct_hgraph  *)(hgraph.extra))->dist[i]; 

 }

 printf("\nMAXDIST=%d\n",maxdist);


 return ;

 }
int   main ( int   argc ,char    *  argv [ ] ) 
 {
hipGetDeviceProperties(&prop0,0); 
alloc_sync_array();


 if( argc!=2  )
{

 printf("error: exec filename");


 return 1;

 }

 SSSP(argv[1]);


 }
