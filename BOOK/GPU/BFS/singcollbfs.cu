#include "hip/hip_runtime.h"

 #include "singcollbfs.h"
hipDeviceProp_t prop0;
__device__ int   changed =0, coll1cnt =0, coll2cnt =0;
;
 int   hchanged ;


 __global__ void   relaxgraph ( GGraph  graph ,GColl coll1,GColl coll2,int   val ,int FALCX) 
 {
int id;
 if(threadIdx.x+blockDim.x*blockIdx.x>=(*(coll1.size)))return; 
id=coll1.ptr[threadIdx.x+blockDim.x*blockIdx.x];
int p=id;
if( id < graph.npoints){

 int falcft0=graph.index[id+1]-graph.index[id];
int falcft1=graph.index[id];
for(int falcft2=0;falcft2<falcft0;falcft2++){
int ut0=2*(falcft1+falcft2);
 int ut1=graph.edges[ut0].ipe;
int ut2=graph.edges[ut0+1].ipe;
int t=ut1;
if( ((struct struct_hgraph  *)(graph.extra))->dist[ut1]>(val+1) )
{

 ((struct struct_hgraph  *)(graph.extra))->dist[ut1]=val+1; 

 int falcvt1;

falcvt1=GADD(&(coll2.size[0]),1);
coll2.ptr[falcvt1]=t; 

 changed=1; 

 }

 }

 }

}
__global__ void   reset ( GGraph  graph ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int t=id;
if( id < graph.npoints){

 ((struct struct_hgraph  *)(graph.extra))->dist[id]=1234567890; 

 }

}
void   SSSP ( char    *  name ) 
 {

 HGraph  hgraph ;

 

 GGraph graph;


  int   xx =0, temp =0;


 hgraph.read2(name);

int hosthgraph=0;
alloc_extra_hgraph(hgraph,hosthgraph,hgraph.npoints);
; 

 hgraph.cloneGPU(graph,0 );
int graphflag=0;

alloc_extra_graph(graph,graphflag,graph.npoints);

int TPB0=findthreadsperblock(prop0);

int graphpointkernelblocks=findblocksize(graph,graph.npoints,TPB0);

int graphedgekernelblocks=findblocksize(graph,graph.nedges,TPB0);
copygraphcurrentsize(graph);
hipSetDevice(0);


 thrust::device_vector<int> coll1temp(graph.npoints);
GColl coll1;
  coll1.ptr=thrust::raw_pointer_cast(&coll1temp[0]);
alloccoll1size(coll1);
thrust::device_vector<int> coll2temp(graph.npoints);
GColl coll2;
  coll2.ptr=thrust::raw_pointer_cast(&coll2temp[0]);
alloccoll2size(coll2);
thrust::device_vector<int> coll3temp(graph.npoints);
GColl coll3;
  coll3.ptr=thrust::raw_pointer_cast(&coll3temp[0]);
alloccoll3size(coll3);


 int   falcvt2;
falcvt2=0;
struct struct_hgraph  temp1;
 hipMemcpy(&temp1,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
if(hipMemcpy(&(temp1.dist[0]),&(falcvt2),sizeof(int ),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 1");

 ((struct struct_hgraph  *)(hgraph.extra))->dist[0]=0; 

  double   rt1 =rtclock();


 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
reset<<<graphpointkernelblocks,TPB0>>>(graph,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);


 int falcvt3;
hipMemcpy(&falcvt3,&(coll1.size[0]),sizeof(int),hipMemcpyDeviceToHost);
coll1temp[ falcvt3++]=0;
hipMemcpy(&(coll1.size[0]),&falcvt3,sizeof(int),hipMemcpyHostToDevice); 

 int falcvt4;
hipMemcpy(&falcvt4,&(coll1.size[0]),sizeof(int),hipMemcpyDeviceToHost);
relaxgraph<<<(falcvt4)/prop0.maxThreadsPerBlock+1,prop0.maxThreadsPerBlock>>>(graph,coll1,coll2,xx,0);
hipDeviceSynchronize();


 
 while(1)  { 

 int   falcvt5;
falcvt5=0;
if(hipMemcpyToSymbol(HIP_SYMBOL(changed),&(falcvt5),sizeof(int ),0,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 2");

 coll3.ptr=coll1.ptr;


 coll1.ptr=coll2.ptr;


 coll2.ptr=coll3.ptr;


 hipMemcpy(&temp,coll2.size,sizeof(int ),hipMemcpyDeviceToHost);


 hipMemcpy(coll1.size,&temp,sizeof(int),hipMemcpyHostToDevice);


 temp=0; 

 hipMemcpy(coll2.size,&temp,sizeof(int),hipMemcpyHostToDevice);


 xx++; 

 int falcvt6;
hipMemcpy(&falcvt6,&(coll1.size[0]),sizeof(int),hipMemcpyDeviceToHost);
relaxgraph<<<(falcvt6)/prop0.maxThreadsPerBlock+1,prop0.maxThreadsPerBlock>>>(graph,coll1,coll2,xx,0);
hipDeviceSynchronize();


 int   falcvt7;
if(hipMemcpyFromSymbol(&(falcvt7),changed,sizeof(int ),0,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 3");
if(falcvt7==0)break;

 }

  double   rt2 =rtclock();


 printf(" TIME %f \n",(rt2-rt1)*1000);


  int   maxdist =0;


 struct struct_hgraph  temp2;/*err*/
hipMemcpy(&temp2,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
if(hipMemcpy((((struct struct_hgraph  *)(hgraph.extra))->dist),(temp2.dist),sizeof(int)*hgraph.npoints,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 4");

 for (int   i =0;i<hgraph.npoints;i++) {

 if( maxdist<((struct struct_hgraph  *)(hgraph.extra))->dist[i]  )
maxdist=((struct struct_hgraph  *)(hgraph.extra))->dist[i]; 

 }

 printf("\nMAXDIST=%d\n",maxdist);


 }
int   main ( int   argc ,char    *  argv [ ] ) 
 {
hipGetDeviceProperties(&prop0,0); 
alloc_sync_array();


 if( argc!=2  )
{

 printf("exec inputgraph\n");


 return 1;

 }

 SSSP(argv[1]);


 }
