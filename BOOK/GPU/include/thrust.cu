//#include <thrust/device_vector.h>
//#include <thrust/host_vector.h>

#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
struct dev_vec{
int pos;
int size;
int  D_Vec[2048];
//int  *D_Vec;
};
__device__ struct dev_vec  create_coll_device( int n){
struct dev_vec t1;
t1.pos=2046;
t1.size=0;
return t1;
}
__device__ int get_elem_coll_device(struct dev_vec &t1,int &val){
t1.size--;
if(t1.size<=2046 && t1.size>=0){ val=t1.D_Vec[t1.size];return val;}
else{ return 0;}
}
__device__ int add_elem_coll_device(struct dev_vec t1,int pos,int val){
if(t1.size>pos){ t1.D_Vec[t1.pos]=val;return 1;}
if(t1.size==pos && pos<t1.pos){
t1.D_Vec[pos]=val;
t1.size++;
return  1;
}
return 0;
}
__device__ void add_coll_device(struct dev_vec &t1,int value){
if(t1.size>t1.pos){
//printf("EXCESS\n");
//t1.size--;
return;
}
t1.D_Vec[t1.size]=value;
t1.size++;
}

/*main(){
test<<<1,1>>>();
cudaDeviceSynchronize();
}*/



